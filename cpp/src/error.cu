/***************************************************************************
 *            error.cpp
 *
 *  Copyright  2021  Mirco De Marchi
 *
 ****************************************************************************/


#include "error.cuh"


namespace cuda_error {

void cudaErrorHandler(hipError_t error,
                      const char* error_message,
                      const char* file,
                      int         line,
                      const char* func_name) {
    if (hipSuccess != error) {
        std::cerr << "\nCUDA error\n" << file << "(" << line << ")"
                  << " [ " << func_name << " ] : " << error_message
                  << " -> " << hipGetErrorString(error)
                  << "(" << static_cast<int>(error) << ")\n"
                  << std::endl;
        assert(false);                                                  //NOLINT
        std::atexit(reinterpret_cast<void(*)()>(hipDeviceReset));
        std::exit(EXIT_FAILURE);
    }
}

void getLastCudaError(const char* file, int line, const char* func_name) {
    cudaErrorHandler(hipGetLastError(), "", file, line, func_name);
}

void safe_call(hipError_t error,
               const char* file,
               int         line,
               const char* func_name) {
    cudaErrorHandler(error, "", file, line, func_name);
}


} // namespace cuda_error